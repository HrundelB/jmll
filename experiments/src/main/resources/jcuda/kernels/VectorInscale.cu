
#include <hip/hip_runtime.h>
extern "C"
__global__ void fSigmoid(float *original, int size) {
  const int X = gridDim.x;
  const int index = gridDim.y * X * threadIdx.x + X * blockIdx.y + blockIdx.x;

  if(index < size) {
    original[index] = 1.f / (1.f + expf(-original[index]));
  }
}

extern "C"
__global__ void fExp(float *original, int size) {
  const int X = gridDim.x;
  const int index = gridDim.y * X * threadIdx.x + X * blockIdx.y + blockIdx.x;

  if(index < size) {
    original[index] = expf(original[index]);
  }
}
